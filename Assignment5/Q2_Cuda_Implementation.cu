#include "hip/hip_runtime.h"
%%cu

// CUDA Implementation goes here!
#include <stdio.h>
#include <stdio.h>
#include <time.h>
#include "hip/hip_runtime.h"
#define MaxThreads 1024
#define CHK(call) {hipError_t err = call; if (err != hipSuccess) { printf("Error%d: %s:%d\n",err,__FILE__,__LINE__); printf(hipGetErrorString(err)); hipDeviceReset(); exit(1);}}


__global__ void vector_init(double *arr, int size);
int main(){
    clock_t start, end;
    double *a, *d_a;
    const int n = 10000000;
    int block_number = (int) (n-1)/ MaxThreads + 1;
    a = (double*) malloc(n * sizeof(double));
    if (NULL == a){
        printf("enable to allocate memory");
        exit(0);
    }
    CHK(hipMalloc(&d_a, n *sizeof(double));)
    start = clock();
    vector_init<<<block_number, MaxThreads>>>(d_a, n);
    hipMemcpy(a, d_a, n * sizeof(double), hipMemcpyDeviceToHost);
    end = clock();
    double total_time = ((double) (end - start)) / CLOCKS_PER_SEC; 
    printf("The total time to execute the serial code is %.3f \n", total_time );
    for (int i=0; i<5; i++){
        printf("a[%d]: %.7f \n", i, a[i]);
    }
    printf("....\n");
    for (int i=n-5; i<n; i++){
        printf("a[%d]: %.7f \n", i, a[i]);
    }


}
__global__ void vector_init(double *arr, int size){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < size){
        double val = (double) i / size;
        arr[i] = val;
    }
}

/*
The total time to execute the serial code is 0.053 
a[0]: 0.0000000 
a[1]: 0.0000001 
a[2]: 0.0000002 
a[3]: 0.0000003 
a[4]: 0.0000004 
....
a[9999995]: 0.9999995 
a[9999996]: 0.9999996 
a[9999997]: 0.9999997 
a[9999998]: 0.9999998 
a[9999999]: 0.9999999 
*/