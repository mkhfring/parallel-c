#include "hip/hip_runtime.h"
%%cuda --name Julia_set.cu --compile true

#include <stdio.h>
#include "EasyBMP.h"
#include "EasyBMP.cu"

//Complex number definition
struct Complex {	// typedef is not required for C++
	float x; 				// real part is represented on x-axis in output image
	float y; 				// imaginary part is represented by y-axis in output image
};

//Function declarations
void compute_julia(const char*, int, int);
void save_image(uchar4*, const char*, int, int);
__device__ Complex add(Complex, Complex);
__device__ Complex mul(Complex, Complex);
__device__ float mag(Complex);

//main function
int main(void) {
	char* name = "test.bmp";
	compute_julia(name, 3000, 3000);	//width x height
	printf("Finished creating %s.\n", name);
	return 0;
}

// serial implementation of Julia set
__global__ void compute_julia_cuda(int width, int height, float x_min, float y_min, float x_incr, float y_incr, uchar4 *p, int infinity, int max_iterations, Complex c1){
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	if (col < width && row < height){		
		Complex z;
		z.x = x_min + col * x_incr;
		z.y = y_min + row * y_incr;
		int n = 0;
		do{
			z = add(mul(z, z), c1);								// z = z^2 + c
		} while (mag(z) < infinity && n++ < max_iterations);	// keep looping until z->infinity or we reach max_iterations
		if (n == max_iterations) {								// if we reach max_iterations before z reaches infinity, pixel is black 
			p[col + row * width] = { 0,0,0,0 };
		} else {												// if z reaches infinity, pixel color is based on how long it takes z to go to infinity
			unsigned char hue = (unsigned char)(255 * sqrt((float)n / max_iterations));
			p[col + row * width] = { hue,hue,hue,255 };
		}
				
	}

}


void compute_julia(const char* filename, int width, int height) {
	//create output image
	uchar4 *pixels = (uchar4*)malloc(width * height * sizeof(uchar4));	//uchar4 is a CUDA type representing a vector of 4 chars

	//PROBLEM SETTINGS (marked by '******')
	// **** Accuracy ****: lower values give less accuracy but faster performance
	int max_iterations = 400;
	int infinity = 20;													//used to check if z goes towards infinity

	// ***** Shape ****: other values produce different patterns. See https://en.wikipedia.org/wiki/Julia_set
	Complex c = { 0.285, 0.01 }; 										//the constant in z = z^2 + c

	// ***** Size ****: higher w means smaller size
	float w = 4;
	float h = w * height / width;										//preserve aspect ratio

	// LIMITS for each pixel
	float x_min = -w / 2, y_min = -h / 2;
	float x_incr = w / width, y_incr = h / height;
	
	//****************************************************
	//REQ: Parallelize the following for loop using CUDA 
	//****************************************************
	uchar4 *d_pixels;
	int size = width * height * sizeof(uchar4);
	hipMalloc(&d_pixels, size);
	hipMemcpy(d_pixels, pixels, size, hipMemcpyHostToDevice);
	dim3 grid, block;
	block.x = 32;	
	block.y = 32;
	grid.x = (width -1) / block.x + 1;
	grid.y = (width -1) / block.y + 1;
	compute_julia_cuda<<<grid, block>>>(width, height, x_min, y_min, x_incr, y_incr, d_pixels, infinity, max_iterations, c);
	hipDeviceSynchronize();

	hipMemcpy(pixels, d_pixels, size, hipMemcpyDeviceToHost);

	//Write output image to a file (DO NOT parallelize this function)
	save_image(pixels, filename, width, height);

	//free memory
	hipFree(d_pixels);
	free(pixels);
}

void save_image(uchar4* pixels, const char* filename, int width, int height) {
	BMP output;
	output.SetSize(width, height);
	output.SetBitDepth(24);
	// save each pixel to output image
	for (int row = 0; row < height; row++) {
		for (int col = 0; col < width; col++) {
			uchar4 color = pixels[col + row * width];
			output(col, row)->Red = color.x;
			output(col, row)->Green = color.y;
			output(col, row)->Blue = color.z;
		}
	}
	output.WriteToFile(filename);
}

__device__ Complex add(Complex c1, Complex c2) {
	return{ c1.x + c2.x, c1.y + c2.y };
}

__device__ Complex mul(Complex c1, Complex c2) {
	return{ c1.x * c2.x - c1.y * c2.y, c1.x * c2.y + c2.x * c1.y };
}

__device__ float mag(Complex c) {
	return (float)sqrt((double)(c.x * c.x + c.y * c.y));
}

